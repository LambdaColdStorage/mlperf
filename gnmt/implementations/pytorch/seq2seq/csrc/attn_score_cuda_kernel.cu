#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/AccumulateType.h>
#include <THC/THC.h>

#define ASSERT_INT4_ALIGNED(PTR) \
    AT_ASSERTM(is_aligned<int4>(PTR), "Tensor is not int4 aligned")

template<class T>
bool
is_aligned(const void * ptr) noexcept {
    auto iptr = reinterpret_cast<std::uintptr_t>(ptr);
    return !(iptr % alignof(T));
}

/** Each block process TILE_Q*TILE_K*hidden volumn. */
template <int TILE, typename scalar_t, typename accscalar_t, typename outscalar_t>
__global__ void
cunn_AttnScoreForward(
    outscalar_t *output,
    const scalar_t* __restrict__ attn_query,
    const scalar_t* __restrict__ attn_keys,
    const scalar_t* __restrict__ bias,
    const scalar_t* __restrict__ linear_attn,
    int t_q,
    int t_k,
    int hidden) {
    
    extern __shared__ unsigned char smem[];
    auto tmp_q = reinterpret_cast<scalar_t*>(smem);
    auto tmp_k = tmp_q + TILE * blockDim.x;
    auto tmp_b = tmp_k + TILE * blockDim.x;
    auto tmp_l = tmp_b + blockDim.x;
    auto tmp_o = reinterpret_cast<accscalar_t*>(tmp_l + blockDim.x);

    int batch_id = blockIdx.x;
    int q_start = blockIdx.y * TILE;
    int k_start = blockIdx.z * TILE;
    
    attn_query += batch_id*t_q*hidden + q_start*hidden;
    attn_keys += batch_id*t_k*hidden + k_start*hidden;
    output += batch_id*t_q*t_k;

    // initialize intermediate result
    #pragma unroll
    for (int i = 0; i < TILE; i++)
        #pragma unroll
        for (int j = 0; j < TILE; j++)
            tmp_o[i*TILE*blockDim.x+j*blockDim.x+threadIdx.x] = 0;

    // ilpReduce
    int offset = threadIdx.x;
    int last = hidden % blockDim.x;

    // ilpReduce on regular data
    for (; offset < hidden - last; offset += blockDim.x) {
        // prolog: load query slices to shared memory
        for (int i = 0; i < t_q - q_start && i < TILE; i++)
            tmp_q[i*blockDim.x+threadIdx.x] = attn_query[i*hidden+offset];

        // prolog: load key slices to shared memory
        for (int i = 0; i < t_k - k_start && i < TILE; i++)
            tmp_k[i*blockDim.x+threadIdx.x] = attn_keys[i*hidden+offset];

        // prolog: load bias and linear_attn slices to shared memory
        tmp_b[threadIdx.x] = bias[offset];
        tmp_l[threadIdx.x] = linear_attn[offset];

        // main loop
        for (int i = 0; i < t_q - q_start && i < TILE; i++) {
            for (int j = 0; j < t_k - k_start && j < TILE; j++) {
                accscalar_t s = static_cast<accscalar_t>(
                    tmp_q[i*blockDim.x+threadIdx.x] +
                    tmp_k[j*blockDim.x+threadIdx.x] +
                    tmp_b[threadIdx.x]);
                tmp_o[i*TILE*blockDim.x+j*blockDim.x+threadIdx.x] += tanhf(s) * tmp_l[threadIdx.x];
            }
        }
    }

    // ilpReduce on boundary
    for (; offset < hidden; offset += blockDim.x) {
        // prolog: load query slices to shared memory
        for (int i = 0; i < t_q - q_start && i < TILE; i++)
            tmp_q[i*blockDim.x+threadIdx.x] = attn_query[i*hidden+offset];

        // prolog: load key slices to shared memory
        for (int i = 0; i < t_k - k_start && i < TILE; i++)
            tmp_k[i*blockDim.x+threadIdx.x] = attn_keys[i*hidden+offset];

        // prolog: load bias and linear_attn slices to shared memory
        tmp_b[threadIdx.x] = bias[offset];
        tmp_l[threadIdx.x] = linear_attn[offset];

        // main loop
        for (int i = 0; i < t_q - q_start && i < TILE; i++) {
            for (int j = 0; j < t_k - k_start && j < TILE; j++) {
                accscalar_t s = static_cast<accscalar_t>(
                    tmp_q[i*blockDim.x+threadIdx.x] +
                    tmp_k[j*blockDim.x+threadIdx.x] +
                    tmp_b[threadIdx.x]);
                tmp_o[i*TILE*blockDim.x+j*blockDim.x+threadIdx.x] += tanhf(s) * tmp_l[threadIdx.x];
            }
        }
    }

    // blockReduce
    __syncthreads();

    // First warp will perform per-warp reductions for the remaining warps
    uint32_t mask = (((uint64_t)1) << (blockDim.x / 32)) - 1;
    if (threadIdx.x < 32) {
        int lane = threadIdx.x % 32;
        if (lane < blockDim.x / 32) {
            for (int i = 0; i < t_q - q_start && i < TILE; i++) {
                for (int j = 0; j < t_k - k_start && j < TILE; j++) {
                    accscalar_t warpVal = static_cast<accscalar_t>(0);
                    #pragma unroll
                    for (int k = 0; k < 32; ++k) {
                        warpVal += tmp_o[i*TILE*blockDim.x+j*blockDim.x+lane*32+k];
                    }
                    __syncwarp(mask);
                    tmp_o[i*TILE*blockDim.x+j*blockDim.x+lane] = warpVal;
                }
            }
        }
    }

    __syncthreads();

    // First thread will perform a reduction of the above per-warp reductions
    if (threadIdx.x == 0) {
        for (int i = 0; i < t_q - q_start && i < TILE; i++) {
            for (int j = 0; j < t_k - k_start && j < TILE; j++) {
                accscalar_t blockVal = static_cast<accscalar_t>(0);
                for (int k = 0; k < blockDim.x / 32; ++k) {
                    blockVal += tmp_o[i*TILE*blockDim.x+j*blockDim.x+k];
                }
                output[(i+q_start)*t_k+(j+k_start)] = static_cast<outscalar_t>(blockVal);
            }
        }
    }

    // Sync and broadcast
    __syncthreads();
}

at::Tensor attn_score_forward_cuda(
    const at::Tensor &attn_query,
    const at::Tensor &attn_keys,
    const at::Tensor &bias,
    const at::Tensor &linear_attn) {
    int batch_sz = attn_query.size(0);
    int t_q = attn_query.size(1);
    int t_k = attn_keys.size(1);
    int hidden = attn_query.size(2);

    at::Tensor output = at::empty({batch_sz, t_q, t_k}, attn_query.options());

    const int TILE = 4;
    int grid_x = batch_sz;
    int grid_y = (t_q + TILE - 1) / TILE;
    int grid_z = (t_k + TILE - 1) / TILE;

    // Each block process TILE_Q*TILE_K*hidden volumn. 
    dim3 block(128);
    dim3 grid(grid_x, grid_y, grid_z);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // Each block load (TILE_Q+TILE_K)*block.x volumn each time
    // Each block load block.x volumn bias and linear_attn
    // Each thread reserve its local results for intra block reduction
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(attn_query.scalar_type(), "attn_score_fprop", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;
        cunn_AttnScoreForward<TILE, scalar_t, accscalar_t, scalar_t>
        <<<grid, block, (2*TILE+2)*block.x * sizeof(scalar_t)+
            block.x * TILE * TILE * sizeof(accscalar_t), stream>>>(
            output.data<scalar_t>(), attn_query.data<scalar_t>(),
            attn_keys.data<scalar_t>(), bias.data<scalar_t>(),
            linear_attn.data<scalar_t>(), t_q, t_k, hidden
        );
    });

    THCudaCheck(hipGetLastError());
	return output;
}

// Extends cuda/include/hip/hip_vector_types.h
struct __builtin_align__(16) float8 {
    float x0, x1, x2, x3, x4, x5, x6, x7;
};
typedef struct float8 float8;

// Extends torch/include/ATen/AccumulateType.h
template <typename T, typename U>
struct VectorType {};

#if defined(__HIPCC__) || defined(__HIPCC__)
template <> struct VectorType<half, float> { using type = float8;  };
#endif
template <> struct VectorType<at::Half, float> { using type = float8;  };
template <> struct VectorType<float, float>    { using type = float4;  };
template <> struct VectorType<double, double>  { using type = double2; };

template<typename T, typename U>
using vec_type = typename VectorType<T, U>::type;

// Convert int4 data to corresponding to vector type
void __device__ __inline__ int4ToVector(float8 *dst, int4 *src) {
    at::Half *src_t = reinterpret_cast<at::Half *>(src);
    dst->x0 = static_cast<float>(src_t[0]);
    dst->x1 = static_cast<float>(src_t[1]);
    dst->x2 = static_cast<float>(src_t[2]);
    dst->x3 = static_cast<float>(src_t[3]);
    dst->x4 = static_cast<float>(src_t[4]);
    dst->x5 = static_cast<float>(src_t[5]);
    dst->x6 = static_cast<float>(src_t[6]);
    dst->x7 = static_cast<float>(src_t[7]);
}
void __device__ __inline__ int4ToVector(float4 *dst, int4 *src) {
    float4 *src_t = reinterpret_cast<float4 *>(src);
    *dst = *src_t;
}
void __device__ __inline__ int4ToVector(double2 *dst, int4 *src) {
    double2 *src_t = reinterpret_cast<double2 *>(src);
    *dst = *src_t;
}

// Convert vector type to int4
void __device__ __inline__ vectorToInt4(int4 *dst, float8 *src) {
    at::Half *dst_t = reinterpret_cast<at::Half *>(dst);
    dst_t[0] = static_cast<at::Half>(src->x0);
    dst_t[1] = static_cast<at::Half>(src->x1);
    dst_t[2] = static_cast<at::Half>(src->x2);
    dst_t[3] = static_cast<at::Half>(src->x3);
    dst_t[4] = static_cast<at::Half>(src->x4);
    dst_t[5] = static_cast<at::Half>(src->x5);
    dst_t[6] = static_cast<at::Half>(src->x6);
    dst_t[7] = static_cast<at::Half>(src->x7);
}
void __device__ __inline__ vectorToInt4(int4 *dst, float4 *src) {
    int4 *src_t = reinterpret_cast<int4 *>(src);
    *dst = *src_t;
}
void __device__ __inline__ vectorToInt4(int4 *dst, double2 *src) {
    int4 *src_t = reinterpret_cast<int4 *>(src);
    *dst = *src_t;
}

/**
 * Each block process BZ*t_q*t_k*LEN volumn.
 */
template <int THREADS, int ILP, int LEN, int TILE, int BZ, typename scalar_t, typename accscalar_t, typename vector_t, typename outscalar_t>
__global__ void
cunn_AttnScoreBackward(
    outscalar_t *grad_query,
    outscalar_t *grad_key,
    outscalar_t *grad_biases,
    outscalar_t *grad_lins,
    const scalar_t* __restrict__ grad_output,
    const scalar_t* __restrict__ attn_query,
    const scalar_t* __restrict__ attn_key,
    const scalar_t* __restrict__ bias,
    const scalar_t* __restrict__ linear_attn,
    int batch_sz,
    int t_q,
    int t_k,
    int hidden) {

    // common parameter check
    static_assert((LEN > 1) & !(LEN & (LEN - 1)), "LEN should be power of 2 for faster mod.");
    static_assert((TILE > 1) & !(TILE & (TILE - 1)), "TILE should be power of 2 for faster round down.");
    static_assert((LEN/ILP > 1) & !(LEN/ILP & (LEN/ILP - 1)), "LEN/ILP should be power of 2 for faster mod.");
    static_assert(TILE*TILE*LEN/ILP%THREADS == 0, "Tailing of tile is not expected.");
    static_assert(TILE*LEN == ILP*THREADS, "Expect threads process a 2D slice of one TILE each time for better performance.");
    static_assert(TILE % ILP == 0, "Expect gradients w.r.t. output can use int4.");

    // calculate rounded up/down bounday
    int t_kd = t_k & ~(TILE - 1);
    int t_qu = (t_q + TILE - 1) / TILE * TILE;
    int t_ku = (t_k + TILE - 1) / TILE * TILE;

    // assign shared memory address
    // keep input key as scalar_t to reduce shared memory usage
    extern __shared__ unsigned char smem[];
    auto tmp_qk = reinterpret_cast<accscalar_t*>(smem);
    auto tmp_gk = tmp_qk + TILE * LEN;
    auto tmp_k = reinterpret_cast<scalar_t*>(tmp_gk + t_ku * LEN);

    // calculate hidden start and batch start
    int tid = threadIdx.x;
    int h_start = blockIdx.x % (hidden / LEN) * LEN;
    int n_start = blockIdx.x / (hidden / LEN) * BZ;
    int h_offset = (tid & (LEN / ILP - 1)) * ILP;

    // update pointers with offset
    grad_output += n_start * t_q * t_k;
    attn_query  += h_start + n_start * t_q * hidden;
    attn_key    += h_start + n_start * t_k * hidden;
    bias        += h_start;
    linear_attn += h_start;
    grad_query  += h_start + n_start * t_q * hidden;
    grad_key    += h_start + n_start * t_k * hidden;
    grad_biases += blockIdx.x * LEN;
    grad_lins   += blockIdx.x * LEN;

    // load bias and linear_attn volume to registers
    // assume one thread process the same hidden id
    static_assert(THREADS % (LEN / ILP) == 0, "Expect one thread process the same hidden index.");
    vector_t tmp_b, tmp_l;
    int4ToVector(&tmp_b, (int4*)(&bias[h_offset]));
    int4ToVector(&tmp_l, (int4*)(&linear_attn[h_offset]));

    // initialize bias and linear_attn gradients to zero
    vector_t tmp_gb = {0}, tmp_gl = {0};

    for (int n=0; n<BZ && n<(batch_sz-n_start); n++) {
        // initialize gradients of key to zero
        // load batch specific key to shared memory
        for (int i=tid*ILP; i<t_kd*LEN; i+=THREADS*ILP) {
            *(int4*)&tmp_k[i] = *(int4*)&attn_key[i/LEN*hidden + (i&(LEN-1))];
            *(vector_t*)&tmp_gk[i] = {0};
        }
        for (int i=t_kd*LEN+tid*ILP; i<t_ku*LEN; i+=THREADS*ILP) {
            if (i/LEN >= t_k)
                *(int4*)&tmp_k[i] = {0};
            else
                *(int4*)&tmp_k[i] = *(int4*)&attn_key[i/LEN*hidden + (i&(LEN-1))];
            *(vector_t*)&tmp_gk[i] = {0};
        }
        __syncthreads();
         
        // loop each tile along query dimension
        for (int tile_q=0; tile_q<t_qu; tile_q+=TILE) {
            // load per thread query of shape ILP to registers
            // initialize gradients of query to zero
            int q_id = tile_q + tid / (LEN / ILP);
            vector_t tmp_q = {0}, tmp_gq = {0};
            if (q_id < t_q)
                int4ToVector(&tmp_q, (int4*)&attn_query[q_id*hidden + h_offset]);

            // loop each tile along key dimension
            for (int tile_k=0; tile_k<t_ku; tile_k+=TILE) {
                // load per thread g_o of shape TILE to registers
                accscalar_t tmp_go[TILE] = {0};
                if (q_id < t_q) {
                    const scalar_t *grad_o = grad_output + q_id * t_k + tile_k;
                    if (tile_k < t_kd) {
                        #pragma unroll
                        for (int i=0; i<TILE/ILP; i++) {
                            int4ToVector(&((vector_t *)tmp_go)[i],
                                (int4*)&grad_o[i*ILP]);
                        }
                    } else {
                        for (int i=0; i<t_k-t_kd; i++) {
                            tmp_go[i] = static_cast<accscalar_t>(grad_o[i]);
                        }
                    }
                }
                __syncthreads();

                // loop each TILE_Q * LEN slice along key dimension
                for (int k=tile_k; k<tile_k+TILE; k++) {
                    // load per thread k and g_k to registers
                    vector_t tmp_k_r;
                    int idx = k * LEN + h_offset;
                    int4ToVector(&tmp_k_r, (int4*)&tmp_k[idx]);
                 
                    accscalar_t t;
                    vector_t g_qk = {0};
                    #pragma unroll
                    for (int i=0; i<ILP; i++) {
                        t = *((accscalar_t *)(&tmp_q)+i) +
                            *((accscalar_t *)(&tmp_k_r)+i) +
                            *((accscalar_t *)(&tmp_b)+i);
                        t = tanhf(t);
                        *((accscalar_t *)(&tmp_gl)+i) += t * tmp_go[k - tile_k];
                        t = *((accscalar_t *)(&tmp_l)+i) * tmp_go[k - tile_k] *
                            (1.f - t * t);
                        *((accscalar_t *)(&tmp_gq)+i) += t;
                        *((accscalar_t *)(&g_qk)+i) = t;
                    }

                    ((vector_t*)tmp_qk)[tid] = g_qk;
                    __syncthreads();

                    // reduce gradients of key, TILE*LEN == THREADS*ILP
                    t = 0;
                    #pragma unroll
                    for (int i=0; i<ILP; i++) {
                        t += tmp_qk[tid + THREADS*i];
                    }
                    tmp_qk[tid] = t;
                    __syncthreads();
                    if (LEN <= 512 && THREADS >= 1024 && tid < 512)
                        tmp_qk[tid] += tmp_qk[tid + 512];
                    __syncthreads();
                    if (LEN <= 256 && THREADS >= 512 && tid < 256)
                        tmp_qk[tid] += tmp_qk[tid + 256];
                    __syncthreads();
                    if (LEN <= 128 && THREADS >= 256 && tid < 128)
                        tmp_qk[tid] += tmp_qk[tid + 128];
                    __syncthreads();
                    if (LEN <= 64 && THREADS >= 128 && tid < 64)
                        tmp_qk[tid] += tmp_qk[tid + 64];
                    __syncthreads();
                    if (LEN <= 32 && tid < 32) {
                        accscalar_t t;
                        #pragma unroll
                        for (int m=32; m>=LEN; m>>=1) {
                            t = tmp_qk[tid] + tmp_qk[tid + m];
                            __syncwarp();
                            tmp_qk[tid] = t;
                        }
                    }
                    __syncthreads();
                    if (tid < LEN) {
                        tmp_gk[k * LEN + tid] += tmp_qk[tid];
                    }
                    __syncthreads();
                }
            }

            // store g_q to global memory
            // accumulate partial g_b using g_q
            if (q_id < t_q) {
                vectorToInt4((int4*)&grad_query[q_id*hidden + h_offset], &tmp_gq);
                #pragma unroll
                for (int i=0; i<ILP; i++) {
                    *((accscalar_t *)(&tmp_gb)+i) += *((accscalar_t *)(&tmp_gq)+i);
                }
            }
        }

        // store g_k to global memory
        for (int i=tid*ILP; i<t_k*LEN; i+=THREADS*ILP) {
            vectorToInt4((int4*)&grad_key[i/LEN*hidden + (i&(LEN-1))],
                (vector_t*)&tmp_gk[i]);
        }

        // update pointer for next batch
        grad_output += t_q * t_k;
        grad_query  += t_q * hidden;
        grad_key    += t_k * hidden;
        attn_query  += t_q * hidden;
        attn_key    += t_k * hidden;
    }

    // reduce partial g_b, g_l
    auto smem_gb = reinterpret_cast<accscalar_t*>(smem);
    auto smem_gl = smem_gb + THREADS * ILP;

    *(vector_t*)&smem_gb[tid * ILP] = tmp_gb;
    *(vector_t*)&smem_gl[tid * ILP] = tmp_gl;
    __syncthreads();

    accscalar_t s = 0, t = 0;
    #pragma unroll
    for (int i=0; i<ILP; i++) {
        s += smem_gb[tid + THREADS*i];
        t += smem_gl[tid + THREADS*i];
    }
    smem_gb[tid] = s;
    smem_gl[tid] = t;
    __syncthreads();
    if (LEN <= 512 && THREADS >= 1024 && tid < 512) {
        smem_gb[tid] += smem_gb[tid + 512];
        smem_gl[tid] += smem_gl[tid + 512];
    }
    __syncthreads();
    if (LEN <= 256 && THREADS >= 512 && tid < 256) {
        smem_gb[tid] += smem_gb[tid + 256];
        smem_gl[tid] += smem_gl[tid + 256];
    }
    __syncthreads();
    if (LEN <= 128 && THREADS >= 256 && tid < 128) {
        smem_gb[tid] += smem_gb[tid + 128];
        smem_gl[tid] += smem_gl[tid + 128];
    }
    __syncthreads();
    if (LEN <= 64 && THREADS >= 128 && tid < 64) {
        smem_gb[tid] += smem_gb[tid + 64];
        smem_gl[tid] += smem_gl[tid + 64];
    }
    __syncthreads();
    if (LEN <= 32 && tid < 32) {
        #pragma unroll
        for (int m=32; m>=LEN; m>>=1) {
            t = smem_gb[tid] + smem_gb[tid + m];
            s = smem_gl[tid] + smem_gl[tid + m];
            __syncwarp();
            smem_gb[tid] = t;
            smem_gl[tid] = s;
        }
    }
    __syncthreads();

    // store per CTA g_b, g_l to global memory
    if (tid < LEN / ILP) {
        vectorToInt4((int4*)&grad_biases[h_offset], (vector_t*)&smem_gb[h_offset]);
        vectorToInt4((int4*)&grad_lins[h_offset], (vector_t*)&smem_gl[h_offset]);
    }
    __syncthreads();
}

std::vector<at::Tensor> attn_score_backward_cuda(
    const at::Tensor &grad_output,
    const at::Tensor &attn_query,
    const at::Tensor &attn_keys,
    const at::Tensor &bias,
    const at::Tensor &linear_attn) {

    int batch_sz = attn_query.size(0);
    int t_q = attn_query.size(1);
    int t_k = attn_keys.size(1);
    int hidden = attn_query.size(2);

    at::Tensor grad_query = at::empty_like(attn_query);
    at::Tensor grad_keys = at::empty_like(attn_keys);

    const int BZ = 2;
    const int THREADS = 128;
    const int ILP = sizeof(int4) / attn_query.element_size();
    const int len = (t_k <= 80) ? 8 * ILP : 4 * ILP;

    assert(hidden % len == 0);

    // Each CTA process BZ*t_q*t_k*len volume
    // Each thread process 1*1*1*int4 a time
    dim3 block(THREADS);
    dim3 grid(((batch_sz+BZ-1)/BZ) * (hidden/len));

    // Allocate per-CTA buffer for future reduction on bias and linear_attn
    at::Tensor grad_biases = at::empty({grid.x, len}, bias.options());
    at::Tensor grad_lins = at::empty({grid.x, len}, linear_attn.options());

    // Check alignment
    ASSERT_INT4_ALIGNED(grad_query.data_ptr());
    ASSERT_INT4_ALIGNED(grad_keys.data_ptr());
    ASSERT_INT4_ALIGNED(grad_biases.data_ptr());
    ASSERT_INT4_ALIGNED(grad_lins.data_ptr());
    ASSERT_INT4_ALIGNED(grad_output.data_ptr());
    ASSERT_INT4_ALIGNED(attn_query.data_ptr());
    ASSERT_INT4_ALIGNED(attn_keys.data_ptr());
    ASSERT_INT4_ALIGNED(bias.data_ptr());
    ASSERT_INT4_ALIGNED(linear_attn.data_ptr());

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if (t_k <= 80) {
        const int TILE = 16;
        const int THREADS_PER_LEN = 8;
        const int LEN = THREADS_PER_LEN * ILP;
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(attn_query.scalar_type(), "attn_score_bprop", [&] {
            using accscalar_t = at::acc_type<scalar_t, true>;
            using vector_t = vec_type<scalar_t, accscalar_t>;
            cunn_AttnScoreBackward<THREADS, sizeof(int4) / sizeof(scalar_t),
                THREADS_PER_LEN * sizeof(int4) / sizeof(scalar_t), TILE, BZ,
                scalar_t, accscalar_t, vector_t, scalar_t>
            <<<grid, block, (TILE + (t_k + TILE - 1) / TILE * TILE) * LEN *
                sizeof(accscalar_t) + (t_k + TILE - 1) / TILE * TILE * LEN *
                sizeof(scalar_t), stream>>>(
                grad_query.data<scalar_t>(), grad_keys.data<scalar_t>(),
                grad_biases.data<scalar_t>(), grad_lins.data<scalar_t>(),
                grad_output.data<scalar_t>(), attn_query.data<scalar_t>(),
                attn_keys.data<scalar_t>(), bias.data<scalar_t>(),
                linear_attn.data<scalar_t>(), batch_sz, t_q, t_k, hidden
            );
        });
    } else {
        const int TILE = 32;
        const int THREADS_PER_LEN = 4;
        const int LEN = THREADS_PER_LEN * ILP;
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(attn_query.scalar_type(), "attn_score_bprop", [&] {
            using accscalar_t = at::acc_type<scalar_t, true>;
            using vector_t = vec_type<scalar_t, accscalar_t>;
            cunn_AttnScoreBackward<THREADS, sizeof(int4) / sizeof(scalar_t),
                THREADS_PER_LEN * sizeof(int4) / sizeof(scalar_t), TILE, BZ,
                scalar_t, accscalar_t, vector_t, scalar_t>
            <<<grid, block, (TILE + (t_k + TILE - 1) / TILE * TILE) * LEN *
                sizeof(accscalar_t) + (t_k + TILE - 1) / TILE * TILE * LEN *
                sizeof(scalar_t), stream>>>(
                grad_query.data<scalar_t>(), grad_keys.data<scalar_t>(),
                grad_biases.data<scalar_t>(), grad_lins.data<scalar_t>(),
                grad_output.data<scalar_t>(), attn_query.data<scalar_t>(),
                attn_keys.data<scalar_t>(), bias.data<scalar_t>(),
                linear_attn.data<scalar_t>(), batch_sz, t_q, t_k, hidden
            );
        });
    }

    // Reduce bias and linear_attn gradients
    at::Tensor grad_bias = at::sum(grad_biases.view({-1, hidden}), 0);
    at::Tensor grad_lin = at::sum(grad_lins.view({-1, hidden}), 0);

    THCudaCheck(hipGetLastError());
	std::vector<at::Tensor> ret = {grad_query, grad_keys, grad_bias, grad_lin};
	return ret;	
}

